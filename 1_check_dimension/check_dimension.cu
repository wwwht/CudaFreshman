#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void checkIndex(void)
{
  printf("threadIdx:(%d,%d,%d) blockIdx:(%d,%d,%d) blockDim:(%d,%d,%d)\
  gridDim(%d,%d,%d)\n",threadIdx.x,threadIdx.y,threadIdx.z,
  blockIdx.x,blockIdx.y,blockIdx.z,blockDim.x,blockDim.y,blockDim.z,
  gridDim.x,gridDim.y,gridDim.z);
}
int main(int argc,char **argv)
{
  int nElem=6;
  dim3 block(3);
  printf("block.x = %d", block.x);
  dim3 grid((nElem+block.x-1)/block.x);
  printf("grid.x %d grid.y %d grid.z %d\n",grid.x,grid.y,grid.z);
  printf("block.x %d block.y %d block.z %d\n",block.x,block.y,block.z);
  // checkIndex<<<grid,block>>>();
  checkIndex<<<1,10>>>();
  hipDeviceReset();
  return 0;
}
