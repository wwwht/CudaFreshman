#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"

// cpu matrix sum 2D
void sumMatrix2D_CPU(float* MatA, float* MatB, float* MatC, const int nx, const int ny)
{
    float* A = MatA;
    float* B = MatB;
    float* C = MatC;
    for (int i = 0; i < ny; i++) {
        for (int j = 0; j < nx; j ++) {
            C[j] = A[j] + B[j];
        }
        A+=nx;
        B+=nx;
        C+=nx;
    }
}

// gpu matrix sum 2D
__global__ void sumMatrix2D_GPU (float * MatA,float * MatB,float * MatC,int nx,int ny)
{
    int ix = threadIdx.x + blockDim.x * blockIdx.x;
    int iy = threadIdx.y + blockDim.y * blockIdx.y;
    int idx = ix + iy * ny;
    if (ix < nx && iy < ny) {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}
//main

int main()
{
    int device = 0;
    initDevice(device);
    int nx = 1<<14;
    int ny = 1<<14;
    int nxy = nx * ny;

    int nBytes = nxy * sizeof(float);

    // Malloc
    float *A_host = (float*)malloc(nBytes);
    float *B_host = (float*)malloc(nBytes);
    float *C_host = (float*)malloc(nBytes);
    float *C_from_gpu = (float*)malloc(nBytes);

    initialData(A_host, nxy);
    initialData(B_host, nxy);

    // cuda Malloc
    float *A_dev = NULL;
    float *B_dev = NULL;
    float *C_dev = NULL;
    CHECK(hipMalloc((void**)&A_dev,nBytes));
    CHECK(hipMalloc((void**)&B_dev,nBytes));
    CHECK(hipMalloc((void**)&C_dev,nBytes));

    CHECK(hipMemcpy(A_dev, A_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(B_dev, B_host, nBytes, hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;

    dim3 block(dimx, dimy);
    dim3 grid((nx-1) / block.x + 1, (ny - 1) / block.y + 1);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // calculate
    sumMatrix2D_GPU<<<grid, block>>>(A_dev, B_dev, C_dev, nx, ny);
    CHECK(hipMemcpy(C_from_gpu, C_dev, nBytes, hipMemcpyDeviceToHost));
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float tm;
    hipEventElapsedTime(&tm,start,stop);
    printf("GPU Execution configuration<<<(%d,%d),(%d,%d)>>> Time elapsed %f ms\n",
        grid.x,grid.y,block.x,block.y,tm);
    
    sumMatrix2D_CPU(A_host, B_host, C_host, nx , ny);
    checkResult(C_host, C_from_gpu, nxy);

    hipFree(A_dev);
    hipFree(B_dev);
    hipFree(C_dev);
    free(A_host);
    free(B_host);
    free(C_host);
    free(C_from_gpu);
    return 0;

}