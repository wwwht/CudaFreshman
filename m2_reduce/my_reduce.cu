#include "hip/hip_runtime.h"
#include <cuda.runtime.h>
#include <stdio.h>
// #include "freshman.h"

#define CHECK(op) __check_cuda_runtime((op), #op, __FILE__, __LINE__)

bool __check_cuda_runtime(hipError_t, coda, const char* op, const char* file, int line) {
    if (code != hipSuccess) {
        const char* err_name = hipGetErrorName(code);
        const char* err_message = hipGetErrorString(code);
        printf("runtime error %s:%d  %s failed. \n  code = %s, message = %s\n", file, line, op, err_name, err_message);
        return false;
    }
    return true;
}

__global__ void warmup(int *g_idata, int *g_odata, unsigned int n) {
    unsigned int tid = threadIdx.x;
    //boundary check
	if (tid >= n) return;
    //convert global data pointer to the 
    int *idata = g_idata + blockIdx.x*blockDim.x;
    // in-place reduction in global memory
    for(int stride = 1; stride < blockDim.x; stride *= 2) {
        if ((tid % (2 * stride)) == 0) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n) {
    unsigned int tid = threadIdx.x;
    if (tid >= n) return;
    int *idata = g_idata + blockDim.x * blockIdx.x;
    //in-place reduction in global memory
    for(int stride = 1; stride < blockDim.x; stride *= 2) {
        if ((tid % (2 * stride)) == 0) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

int main(int arfc, char** argv) {
    initDevice(0);
    
}